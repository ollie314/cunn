
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiMarginCriterion.cu"
#else

void THNN_(MultiMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           bool sizeAverage,
           int p,
           THCTensor *weights,
           real margin)
{
  THCUNN_assertSameGPU_generic(state, 2, input, target);
  input = THCTensor_(newContiguous)(state, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);
  if (input->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);
    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, input->size[0],
        sizeAverage,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, input->size[0],
        sizeAverage,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->nDimension == 2)
  {
    THCTensor *output_ = THCTensor_(newWithSize1d)(state, input->size[0]);  // tmp outupt buffer
    dim3 blocks(input->size[0]);
    dim3 threads(MULTIMARGIN_THREADS);
    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output_),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        input->size[0], input->size[1],
        sizeAverage,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateOutput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output_),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        input->size[0], input->size[1],
        sizeAverage,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
    float sum = THCTensor_(sumall)(state, output_);
    THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
    THCTensor_(free)(state, output_);
  }
  else
  {
    THError("vector or matrix expected");
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

void THNN_(MultiMarginCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *gradInput,
           bool sizeAverage,
           int p,
           THCTensor *weights,
           real margin)
{
  THCUNN_assertSameGPU_generic(state, 3, input, gradInput, target);
  input = THCTensor_(newContiguous)(state, input);
  THCTensor_(resizeAs)(state, gradInput, input);
  if(weights)
    weights = THCTensor_(newContiguous)(state, weights);

  if (input->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, gradInput->size[0],
        sizeAverage,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        1, gradInput->size[0],
        sizeAverage,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else if (input->nDimension == 2)
  {
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTIMARGIN_THREADS);

    if (p == 1)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<1, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        gradInput->size[0], gradInput->size[1],
        sizeAverage,
        margin
      );
    }
    else if (p == 2)
    {
      cunn_MultiMarginCriterion_updateGradInput_kernel<2, real, accreal> <<<blocks,threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        weights ? THCTensor_(data)(state, weights) : NULL,
        gradInput->size[0], gradInput->size[1],
        sizeAverage,
        margin
      );
    }
    THCudaCheck(cudaGetLastError());
  }
  else
  {
    THError("vector or matrix expected");
  }

  THCTensor_(free)(state, input);
  if(weights)
    THCTensor_(free)(state, weights);
}

#endif
