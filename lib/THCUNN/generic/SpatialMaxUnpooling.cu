
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialMaxUnpooling.cu"
#else

void THNN_(SpatialMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU_generic(state, 3, input, output, indices);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, oheight, owidth);
  THCTensor_(zero)(state, output);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, input), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, output));
  THCudaCheck(cudaGetLastError());

  if(input->nDimension == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, oheight, owidth);

  THCTensor_(free)(state, input);

}

void THNN_(SpatialMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU_generic(state, 4, input, gradOutput, indices, gradInput);

  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, gradOutput), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  // clean
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif
