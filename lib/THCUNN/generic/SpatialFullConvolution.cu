
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialFullConvolution.cu"
#else

void THNN_(SpatialFullConvolution_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int adjW, int adjH)
{

  int nInputPlane = THCTensor_(size)(state, weight, 0);
  int nOutputPlane = THCTensor_(size)(state, weight, 1);

  THCUNN_assertSameGPU_generic(state, 6, input, output, weight,
                                 bias, columns, ones);

  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    THArgCheck(input->size[0] == nInputPlane, 2, "input channels and nInputPlane dont match");
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
  } else {
    THArgCheck(input->size[1] == nInputPlane, 2, "input channels and nInputPlane dont match");
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[1] * weight->size[2] * weight->size[3];
    long n = columns->size[1];
    long k = weight->size[0];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 't',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, input_n), n,
        THCTensor_(data)(state, weight), m,
        ScalarConvert<int, real>::to(0),
        THCTensor_(data)(state, columns), n
    );

    // Unpack columns back into input:
    col2im<real, accreal>(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, columns),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, output_n)
    );

    // Do Bias after:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    if (bias) {
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemm(
      #elif defined(THC_REAL_IS_HALF)
      THCudaBlas_Hgemm(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemm(
      #endif
          state,
          't', 'n',
          n_, m_, k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, ones), k_,
          THCTensor_(data)(state, bias), k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, output_n), n_
      );
    }

  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }
}

void THNN_(SpatialFullConvolution_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *gradColumns,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int adjW, int adjH)
{
  int nInputPlane = THCTensor_(size)(state, weight, 0);
  int nOutputPlane = THCTensor_(size)(state, weight, 1);

  THCUNN_assertSameGPU_generic(state, 5, input, gradOutput, weight,
                                 gradColumns, gradInput);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize4d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize2d)(state, gradColumns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradOutput_n),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, gradColumns)
    );


    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[0];
    long n = gradColumns->size[1];
    long k = weight->size[1] * weight->size[2] * weight->size[3];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        'n', 'n',
        n, m, k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradColumns), n,
        THCTensor_(data)(state, weight), k,
        ScalarConvert<int, real>::to(0),
        THCTensor_(data)(state, gradInput_n), n
    );
  }


  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }
}


void THNN_(SpatialFullConvolution_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *columns,
           THCTensor *ones,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int adjW, int adjH,
           real scale)
{
  int nInputPlane = THCTensor_(size)(state, gradWeight, 0);
  int nOutputPlane = THCTensor_(size)(state, gradWeight, 1);

  THCUNN_assertSameGPU_generic(state, 6, input, gradOutput, gradWeight,
                                 gradBias, columns, ones);

  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCTensor_(resize4d)(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW + adjW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH + adjH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Define a buffer of ones, for bias accumulation
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCTensor_(resize2d)(state, ones, outputHeight, outputWidth);
    THCTensor_(fill)(state, ones, ScalarConvert<int, real>::to(1));
  }

  // Resize temporary columns
  THCTensor_(resize2d)(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, gradOutput_n),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long n = columns->size[0];   // nOutputPlane * kh * kw
    long m = input_n->size[0];   // nInputPlane
    long k = columns->size[1];   // inputHeight * inputWidth

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    #ifdef THC_REAL_IS_FLOAT
    THCudaBlas_Sgemm(
    #elif defined(THC_REAL_IS_HALF)
    THCudaBlas_Hgemm(
    #elif defined(THC_REAL_IS_DOUBLE)
    THCudaBlas_Dgemm(
    #endif
        state,
        't', 'n',
        n, m, k,
        scale,
        THCTensor_(data)(state, columns), k,
        THCTensor_(data)(state, input_n), k,
        ScalarConvert<int, real>::to(1),
        THCTensor_(data)(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long k_ = outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    if (gradBias) {
      #if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
      #ifdef THC_REAL_IS_FLOAT
      THCudaBlas_Sgemv(
      #elif defined(THC_REAL_IS_DOUBLE)
      THCudaBlas_Dgemv(
      #endif
          state,
          't',
          k_, m_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), 1,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), 1
      );
      #endif
      #ifdef THC_REAL_IS_HALF
      THCudaBlas_Hgemm(
          state,
          't', 'n',
          m_, 1, k_,
          scale,
          THCTensor_(data)(state, gradOutput_n), k_,
          THCTensor_(data)(state, ones), k_,
          ScalarConvert<int, real>::to(1),
          THCTensor_(data)(state, gradBias), m_
      );
      #endif
    }
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize
  if (batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }
}

#endif
