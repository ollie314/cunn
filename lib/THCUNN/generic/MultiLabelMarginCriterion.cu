
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiLabelMarginCriterion.cu"
#else

void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           bool sizeaverage)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, input);

  if(input->nDimension == 1)
  {
    THCTensor_(resize1d)(state, output, 1);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal> <<<blocks,threads>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, input->size[0],
        sizeaverage
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->nDimension == 2)
  {
    THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, input->size[0]);

    dim3 blocks(input->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal> <<<blocks,threads>>>(
        THCTensor_(data)(state, output_tmp),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        input->size[0], input->size[1],
        sizeaverage
        );
    THCudaCheck(cudaGetLastError());
    THCTensor_(resize1d)(state, output, 1);
    THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(THCTensor_(sumall)(state, output_tmp)));
    THCTensor_(free)(state, output_tmp);
  }
  else
    THError("vector or matrix expected");

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradInput,
            THCTensor *istarget,
            bool sizeaverage)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, gradInput, input);

  if(gradInput->nDimension == 1)
  {
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal> <<<blocks,threads>>>(THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, gradInput->size[0],
        sizeaverage);

  }
  else if(gradInput->nDimension == 2)
  {
    dim3 blocks(gradInput->size[0]);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal> <<<blocks,threads>>>(THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        gradInput->size[0], gradInput->size[1],
        sizeaverage);
  }
  else
    THError("vector or matrix expected");

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

#endif
